#include "a100_common_fwd.cuh"
#include "a100_common_bwd.cuh"
#include "pyutils/torch_helpers.cuh"
#include <ATen/cuda/HIPContext.h>
#include <iostream>
#include "static_switch.h"
#include "fa_switch.h"

namespace fa_a100 {

using namespace kittens;

std::vector<torch::Tensor>
attention_forward(torch::Tensor q, torch::Tensor k, torch::Tensor v, bool causal, float sm_scale)
{
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);

    auto batch = static_cast<const uint>(q.size(0));
    auto seqlen_q = static_cast<const uint>(q.size(2));
    auto seqlen_k = static_cast<const uint>(k.size(2));
    auto head_dim = static_cast<const uint>(q.size(3));
    auto is_causal = static_cast<const bool>(causal);
    auto qo_heads = static_cast<const uint>(q.size(1));
    auto kv_heads = static_cast<const uint>(k.size(1));

    // check to see that these dimensions match for all inputs
    TORCH_CHECK(q.size(0) == batch, "Q batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(k.size(0) == batch, "K batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(v.size(0) == batch, "V batch dimension - idx 0 - must match for all inputs");

    TORCH_CHECK(q.size(3) == head_dim, "Q head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(k.size(3) == head_dim, "K head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(v.size(3) == head_dim, "V head dimension - idx 3 - must match for all non-vector inputs");

    TORCH_CHECK(qo_heads >= kv_heads, "QO heads must be greater than or equal to KV heads");
    TORCH_CHECK(qo_heads % kv_heads == 0, "QO heads must be divisible by KV heads");
    TORCH_CHECK(q.size(1) == qo_heads, "QO head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(k.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(v.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");

    bf16* q_ptr = reinterpret_cast<bf16*>(q.data_ptr<c10::BFloat16>());
    bf16* k_ptr = reinterpret_cast<bf16*>(k.data_ptr<c10::BFloat16>());
    bf16* v_ptr = reinterpret_cast<bf16*>(v.data_ptr<c10::BFloat16>());

    // for the returned outputs
    torch::Tensor o = torch::empty({batch, qo_heads, seqlen_q, head_dim}, q.options());

    torch::Tensor l_vec = torch::empty({batch, qo_heads, 1, seqlen_q},
                                        torch::TensorOptions().dtype(torch::kFloat).device(q.device()).memory_format(at::MemoryFormat::Contiguous));


    bf16* o_ptr = reinterpret_cast<bf16*>(o.data_ptr<c10::BFloat16>());
    float* l_ptr = reinterpret_cast<float*>(l_vec.data_ptr<float>());

    HEAD_DIM_SWITCH(head_dim, HEAD_DIM, [&] {
        using ker_tile_dims = fwd_ker_tile_dims<HEAD_DIM>;
        const bool is_even_nm = (seqlen_q % ker_tile_dims::qo_height) == 0 && (seqlen_k % ker_tile_dims::kv_height) == 0;

        BOOL_SWITCH(is_causal, IS_CAUSAL, [&] {
            BOOL_SWITCH(is_even_nm, IS_EVEN_NM, [&] {

                hipDeviceSynchronize();
                auto stream = at::cuda::getCurrentCUDAStream().stream();

                using globals = fwd_globals<HEAD_DIM>;

                using q_global = globals::q_gl;
                using k_global = globals::k_gl;
                using v_global = globals::v_gl;
                using l_global = globals::l_gl;
                using o_global = globals::o_gl;

                q_global qg_arg{q_ptr, batch, qo_heads, seqlen_q, head_dim};
                k_global kg_arg{k_ptr, batch, kv_heads, seqlen_k, head_dim};
                v_global vg_arg{v_ptr, batch, kv_heads, seqlen_k, head_dim};
                l_global lg_arg{l_ptr, batch, qo_heads, 1U, seqlen_q};
                o_global og_arg{o_ptr, batch, qo_heads, seqlen_q, head_dim};

                globals g{qg_arg, kg_arg, vg_arg, lg_arg, og_arg, seqlen_q, seqlen_k, qo_heads, kv_heads, sm_scale};

                int max_smem_size;
                hipDeviceGetAttribute(&max_smem_size, hipDeviceAttributeSharedMemPerBlockOptin, 0);

                int smem_size = g.get_smem_size();

                assert(smem_size <= max_smem_size);

                auto threads = FWD_NUM_WORKERS * kittens::WARP_THREADS;

                int q_blocks = (seqlen_q + (FWD_NUM_WORKERS*ker_tile_dims::qo_height)/2) / (FWD_NUM_WORKERS*ker_tile_dims::qo_height);

                dim3 grid(q_blocks, qo_heads, batch);

                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    fwd_attend_ker<HEAD_DIM), IS_CAUSAL, IS_EVEN_NM>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    smem_size
                );

                fwd_attend_ker<HEAD_DIM, IS_CAUSAL, IS_EVEN_NM><<<grid, threads, smem_size, stream>>>(g);
                CHECK_CUDA_ERROR(hipGetLastError());
                hipStreamSynchronize(stream);
                hipDeviceSynchronize();
            });
        });
    });

    return {o, l_vec};
}

std::vector<torch::Tensor>
attention_backward(torch::Tensor q,
                   torch::Tensor k,
                   torch::Tensor v,
                   torch::Tensor o,
                   torch::Tensor l_vec,
                   torch::Tensor og,
                   bool is_causal,
                   float sm_scale)
{
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(l_vec);
    CHECK_INPUT(o);
    CHECK_INPUT(og);

    auto batch = q.size(0);
    auto seqlen_q = q.size(2);
    auto seqlen_k = k.size(2);
    auto qo_heads = q.size(1);
    auto kv_heads = k.size(1);
    auto head_dim = q.size(3);

    // check to see that these dimensions match for all inputs
    TORCH_CHECK(q.size(0) == batch, "Q  batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(k.size(0) == batch, "K  batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(v.size(0) == batch, "V  batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(l_vec.size(0) == batch, "L  batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(o.size(0) == batch, "O  batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(og.size(0) == batch, "OG batch dimension - idx 0 - must match for all inputs");

    TORCH_CHECK(q.size(2) == seqlen_q, "Q  sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(k.size(2) == seqlen_k, "K  sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(v.size(2) == seqlen_k, "V  sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(l_vec.size(2) == seqlen_q, "L  sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(o.size(2) == seqlen_q, "O  sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(og.size(2) == seqlen_q, "OG sequence length dimension - idx 2 - must match for all inputs");

    TORCH_CHECK(q.size(3) == head_dim, "Q  head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(k.size(3) == head_dim, "K  head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(v.size(3) == head_dim, "V  head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(o.size(3) == head_dim, "O  head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(og.size(3) == head_dim, "OG head dimension - idx 3 - must match for all non-vector inputs");

    TORCH_CHECK(qo_heads >= kv_heads, "Q heads must be greater than or equal to K and V heads");
    TORCH_CHECK(qo_heads % kv_heads == 0, "Q heads must be divisible by KV heads");

    TORCH_CHECK(q.size(1) == qo_heads, "Q  heads dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(l_vec.size(1) == qo_heads, "L  heads dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(o.size(1) == qo_heads, "O  heads dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(og.size(1) == qo_heads, "OG heads dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(k.size(1) == kv_heads, "K  heads dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(v.size(1) == kv_heads, "V  heads dimension - idx 1 - must match for all inputs");

    // Initialize tensors for the gradients
    torch::Tensor qg = torch::zeros_like(q, q.options());
    torch::Tensor kg = torch::zeros_like(k, k.options());
    torch::Tensor vg = torch::zeros_like(v, v.options());
    torch::Tensor d_vec = torch::empty_like(l_vec, l_vec.options());

    bf16* d_q = reinterpret_cast<bf16*>(qg.data_ptr<c10::BFloat16>());
    bf16* d_k = reinterpret_cast<bf16*>(kg.data_ptr<c10::BFloat16>());
    bf16* d_v = reinterpret_cast<bf16*>(vg.data_ptr<c10::BFloat16>());
    bf16* d_o = reinterpret_cast<bf16*>(o.data_ptr<c10::BFloat16>());
    bf16* d_og = reinterpret_cast<bf16*>(og.data_ptr<c10::BFloat16>());
    float* d_l = reinterpret_cast<float*>(l_vec.data_ptr<float>());
    float* d_d = reinterpret_cast<float*>(d_vec.data_ptr<float>());
    bf16* d_qg = reinterpret_cast<bf16*>(qg.data_ptr<c10::BFloat16>());
    bf16* d_kg = reinterpret_cast<bf16*>(kg.data_ptr<c10::BFloat16>());
    bf16* d_vg = reinterpret_cast<bf16*>(vg.data_ptr<c10::BFloat16>());

    HEAD_DIM_SWITCH(head_dim, HEAD_DIM, [&] {
        using ker_tile_dims = bwd_ker_tile_dims<HEAD_DIM>;
        const bool is_even_nm = (seqlen_q % ker_tile_dims::qo_height) == 0 && (seqlen_k % ker_tile_dims::kv_height) == 0;

        BOOL_SWITCH(is_causal, IS_CAUSAL, [&] {
            BOOL_SWITCH(is_even_nm, IS_EVEN_NM, [&] {

                hipDeviceSynchronize();
                auto stream = at::cuda::getCurrentCUDAStream().stream();

                using bwd_prep_globals = bwd_prep_globals<HEAD_DIM>;

                using o_global = bwd_prep_globals::o_gl;
                using og_global = bwd_prep_globals::og_gl;
                using d_global = bwd_prep_globals::d_gl;

                o_global o_arg{d_o, batch, qo_heads, seqlen_q, head_dim};
                og_global og_arg{d_og, batch, qo_heads, seqlen_q, head_dim};
                d_global d_arg{d_d, batch, qo_heads, 1U, seqlen_q};

                bwd_prep_globals pg{o_arg, og_arg, d_arg};

                int max_smem_size;
                hipDeviceGetAttribute(&max_smem_size, hipDeviceAttributeSharedMemPerBlockOptin, 0);

                int smem_size = pg.get_smem_size();

                assert(smem_size <= max_smem_size);

                auto threads = BWD_PREP_NUM_WORKERS * kittens::WARP_THREADS;

                int q_blocks = (seqlen_q + (BWD_PREP_NUM_WORKERS*ker_tile_dims::qo_height)) / (BWD_PREP_NUM_WORKERS*ker_tile_dims::qo_height);

                dim3 grid(q_blocks, qo_heads, batch);

                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    bwd_prep_ker<HEAD_DIM>),
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    smem_size
                );

                bwd_prep_ker<HEAD_DIM><<<grid, threads, smem_size, stream>>>(pg);
                CHECK_CUDA_ERROR(hipGetLastError());
                hipStreamSynchronize(stream);
                hipDeviceSynchronize();

            });
        });
    });

    std::cout << "d_vec: " << d_vec << std::endl;

    return {qg, kg, vg, d_vec};
}

} // namespace fa_a100

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fwd", &fa_a100::attention_forward, "Forward pass");
    m.def("bwd", &fa_a100::attention_backward, "Backward pass");
}